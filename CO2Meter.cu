/*********************************************
UNIVERSIDAD DEL VALLE DE GUATEMALA
CC3056 - Programación de Microprocesadores
Ciclo 2 - 2019

Authors: Gustavo Mendez, Roberto Figueroa, Marco Fuentes
Date: Oct. 27, 2019
File: CO2Meter.cu
Description: Test of measurement model of CO2 in a closed environment, depending on the temperature,
            pressure, volume of room and number of people in it.
**********************************************/
#include <stdio.h> 
#include <stdlib.h> 
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

//CSV Reader 
#include <iostream>
#include <fstream>
using namespace std;

#define N 83000                 //Threads, and length of arrays declared

#define R (8.314f)               //Gas constant
#define ACH (3.5f)               //ACH for a classroom
#define CO2_MASS (44.01f)        //CO2 Molar mass 
#define CO2_ADULT_GAIN (0.0052f) //CO2 gain by an adult, in l/s
#define A304_VOLUME (114.20f)    //m3
#define C114_VOLUME (125.625f)   //m3
#define A304_PERSONS (22.0f)       // no. persons in this classroom
#define C114_PERSONS (16.0f)       // no. persons in this classroom

#define CO2_OUTDOOR (404.0f)       //CO2 outside, in PPM
#define PRESSURE_OUTDOOR (852.0f)  //Pressure outside
#define TEMP_OUTDOOR (24.1f)     //Temperature outside 

/* CO2 model check function */
__global__ void getGasModelKernel1(float *a1, float *b1, float *c1, float *d1)
{
    float currVolume = A304_VOLUME;
    float currPersons = A304_PERSONS;

    int index = threadIdx.x + blockDim.x * blockIdx.x;				
	if (index < N)
	{
		d1[index] = (b1[index] / (c1[index] * (1 + ACH * (index/10)))) * ( ((a1[0] * c1[0])/b1[0]) + ( (((CO2_ADULT_GAIN * currPersons * R)/(CO2_MASS * currVolume)) + ((CO2_OUTDOOR * PRESSURE_OUTDOOR * ACH)/(TEMP_OUTDOOR))) * (index/10)));
	}
    
}

/* CO2 model check function */
__global__ void getGasModelKernel2(float *a2, float *b2, float *c2, float *d2)
{    
    //AVG a[index]
    float currVolume = C114_VOLUME;
    float currPersons = C114_PERSONS;

    int index = threadIdx.x + blockDim.x * blockIdx.x;				
	if (index < N)
	{
		d2[index] = (b2[index] / (c2[index] * (1 + ACH * (index/10)))) * ( ((a2[0] * c2[0])/b2[0]) + ( (((CO2_ADULT_GAIN * currPersons * R)/(CO2_MASS * currVolume)) + ((CO2_OUTDOOR * PRESSURE_OUTDOOR * ACH)/(TEMP_OUTDOOR))) * (index/10)));
	}
    
}

int main(void)
{
    /* Vectors contains:
        - a: real CO2 value, in PPM
        - b: temperature value, in Celsius
        - c: pressure value, in hPa
        - e: CO2 model value, in PPM
    */

    hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
    
    float *a1, *b1, *c1, *d1, *a2, *b2, *c2, *d2; //host copies of a,b,c vectors
    float *d_a1, *d_b1, *d_c1, *d_d1, *d_a2, *d_b2, *d_c2, *d_d2; //device copies of a,b,c etc

    int size = N * sizeof(float); //size of memory that needs to be allocated

    //Allocate space for device copies of a,b,c, e
    hipMalloc((void **)&d_a1, size);
    hipMalloc((void **)&d_b1, size);
    hipMalloc((void **)&d_c1, size);
    hipMalloc((void **)&d_d1, size);
    hipMalloc((void **)&d_a2, size);
    hipMalloc((void **)&d_b2, size);
    hipMalloc((void **)&d_c2, size);
    hipMalloc((void **)&d_d2, size);

    //Allows device to get access to memory
    hipHostAlloc( (void**)&a1, size, hipHostMallocDefault);	 
    hipHostAlloc( (void**)&b1, size, hipHostMallocDefault);
    hipHostAlloc( (void**)&c1, size, hipHostMallocDefault);
    hipHostAlloc( (void**)&d1, size, hipHostMallocDefault);
    hipHostAlloc( (void**)&a2, size, hipHostMallocDefault);	 
    hipHostAlloc( (void**)&b2, size, hipHostMallocDefault);
    hipHostAlloc( (void**)&c2, size, hipHostMallocDefault);
    hipHostAlloc( (void**)&d2, size, hipHostMallocDefault);

    /*Loading CSV
        Room Structure:
            - A304: 1 - 41500
            - C114: 41501 - 83000 
    */
	ifstream ip("finalData.csv");

	if (!ip.is_open())
		std::cout << "ERROR: File Open" << '\n';

	//Gas means CO2 value, in PPM
	string gasString, temperatureString, pressureString;

    int i = 0;
	while (ip.good())
	{
		//Reading by columns
		getline(ip, gasString, ',');
		getline(ip, temperatureString, ',');
		getline(ip, pressureString, '\n');

        //Cast string to float, and saving values in array
		double gasDouble = ::atof(gasString.c_str());
		double tempDouble = ::atof(temperatureString.c_str());
        double pressureDouble = ::atof(pressureString.c_str());

        if(N/2 < 41500) {
            a1[i] = float(gasDouble);
            b1[i] = float(tempDouble);
            c1[i] = float(pressureDouble);
        } else {
            a2[i] = float(gasDouble);
            b2[i] = float(tempDouble);
            c2[i] = float(pressureDouble);
        }

        i++;
				
    }
    
    printf("# OF DATA: %d \n", i - 1);
	ip.close();
    

    //Timing
    struct timeval t1, t2;
    gettimeofday(&t1, 0);

    for(int i=0;i < N;i++)
	{   // loop over data in chunks
	    // stream 1
		hipMemcpyAsync(d_a1, a1, size, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_b1, b1, size, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_c1, c1, size, hipMemcpyHostToDevice, stream1);
		getGasModelKernel1<<<1,N,0,stream1>>>(d_a1, d_b1, d_c1, d_d1);
		hipMemcpyAsync(d1, d_d1, size, hipMemcpyDeviceToHost, stream1);

		//stream 2
		hipMemcpyAsync(d_a2, a2, size, hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(d_b2, b2, size, hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d_c2, c2, size, hipMemcpyHostToDevice, stream2);
        getGasModelKernel2<<<1,N,1,stream2>>>(d_a2, d_b2, d_c2, d_d2);
		hipMemcpyAsync(d2, d_c2, size, hipMemcpyDeviceToHost, stream2);
    }
    
	hipStreamSynchronize(stream1); // wait for stream1 to finish
	hipStreamSynchronize(stream2); // wait for stream2 to finish

    gettimeofday(&t2, 0);
    float time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("EXECUTION TIME:  %5.4f ms \n", time);

    //A304
    //Some values of real CO2 from 3000 to 3010
    printf("MODEL CO2 - A304 = [");
    for (i=25000; i<25010; i++) printf(" %4.3f", d1[i]);
    printf(" ...]\n");

    //Some values of real CO2 from 3000 to 3010
    printf("REAL CO2 - A304= [");
    for (i=25000; i<25010; i++) printf(" %4.3f", a1[i]);
    printf(" ...]\n");
    
    
    //C114
    //Some values of real CO2 from 3000 to 3010
    printf("MODEL CO2 - C114 = [");
    for (i=25000; i<25010; i++) printf(" %4.3f", d2[i]);
    printf(" ...]\n");

    //Some values of real CO2 from 3000 to 3010
    printf("REAL CO2 - C114 = [");
    for (i=25000; i<25010; i++) printf(" %4.3f", a2[i]);
    printf(" ...]\n");


    //Destroying stream used
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
	return 0;
} 